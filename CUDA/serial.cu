#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>

#include <array>
#include <vector>

#include "common.h"

typedef struct
{
  std::vector<std::array<int, 2>> id;
  std::vector<int> particle_id;
  std::vector<std::array<int, 2>> neighbor_bin_id;
} bin_t;

//
//  benchmarking program
//
int main( int argc, char **argv )
{    
    int navg,nabsavg=0;
    double davg,dmin, absmin=1.0, absavg=0.0;

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        printf( "-s <filename> to specify a summary file name\n" );
        printf( "-no turns off all correctness checks and particle output\n");
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    char *sumname = read_string( argc, argv, "-s", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    FILE *fsum = sumname ? fopen ( sumname, "a" ) : NULL;

    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );
    set_size( n );
    init_particles( n, particles );
    
    // Initialize bin and assign neighboring bins
    double bin_size = cutoff;
    double area_size = sqrt( density * n );
    int nbin_1d = floor(area_size/bin_size);
    if(fmod(area_size, bin_size) != 0) nbin_1d++;
    bin_t bin[nbin_1d][nbin_1d];
    for(int row = 0; row < nbin_1d; row++) {
      for(int col = 0; col < nbin_1d; col++) {
        std::array<int, 2> id_temp = {row, col};
        bin[row][col].id.push_back(id_temp);
        if(row == 0) {
	  if(col == 0) {
            std::array<int, 2> temp[3];
            temp[0] = {0,1};
            temp[1] = {1,0};
            temp[2] = {1,1};
            for(int i = 0; i < sizeof(temp)/sizeof(*temp); i++)
	      bin[row][col].neighbor_bin_id.push_back(temp[i]);
	  } else if (col == nbin_1d-1) {
	    std::array<int, 2> temp[3];
            temp[0] = {row+1,col};
            temp[1] = {row,col-1};
            temp[2] = {row+1,col-1};
            for(int i = 0; i < sizeof(temp)/sizeof(*temp); i++)
	      bin[row][col].neighbor_bin_id.push_back(temp[i]);
    	  } else {
	    std::array<int, 2> temp[5];
            temp[0] = {row,col-1};
            temp[1] = {row,col+1};
            temp[2] = {row+1,col-1};
            temp[3] = {row+1,col};
            temp[4] = {row+1,col+1};
            for(int i = 0; i < sizeof(temp)/sizeof(*temp); i++)
	      bin[row][col].neighbor_bin_id.push_back(temp[i]);
    	  } 
	}    
        else if(row == nbin_1d-1) {
	  if(col == 0) {
            std::array<int, 2> temp[3];
            temp[0] = {row-1,0};
            temp[1] = {row-1,1};
            temp[2] = {row,1};
            for(int i = 0; i < sizeof(temp)/sizeof(*temp); i++)
	      bin[row][col].neighbor_bin_id.push_back(temp[i]);
	  } else if (col == nbin_1d-1) {
	    std::array<int, 2> temp[3];
            temp[0] = {row-1,col};
            temp[1] = {row-1,col-1};
            temp[2] = {row,col-1};
            for(int i = 0; i < sizeof(temp)/sizeof(*temp); i++)
	      bin[row][col].neighbor_bin_id.push_back(temp[i]);
    	  } else {
	    std::array<int, 2> temp[5];
            temp[0] = {row,col-1};
            temp[1] = {row,col+1};
            temp[2] = {row-1,col-1};
            temp[3] = {row-1,col};
            temp[4] = {row-1,col+1};
            for(int i = 0; i < sizeof(temp)/sizeof(*temp); i++)
	      bin[row][col].neighbor_bin_id.push_back(temp[i]);
    	  }
	}
	else {
	  if(col == 0) {
            std::array<int, 2> temp[5];
            temp[0] = {row-1,0};
            temp[1] = {row-1,1};
            temp[2] = {row,1};
            temp[3] = {row+1,0};
            temp[4] = {row+1,1};
            for(int i = 0; i < sizeof(temp)/sizeof(*temp); i++)
	      bin[row][col].neighbor_bin_id.push_back(temp[i]);
	  } else if (col == nbin_1d-1) {
	    std::array<int, 2> temp[5];
            temp[0] = {row-1,col};
            temp[1] = {row-1,col-1};
            temp[2] = {row,col-1};
            temp[3] = {row+1,col-1};
            temp[4] = {row+1,col};
            for(int i = 0; i < sizeof(temp)/sizeof(*temp); i++)
	      bin[row][col].neighbor_bin_id.push_back(temp[i]);
    	  } else {
	    std::array<int, 2> temp[8];
            temp[0] = {row,col-1};
            temp[1] = {row,col+1};
            temp[2] = {row-1,col-1};
            temp[3] = {row-1,col};
            temp[4] = {row-1,col+1};
            temp[5] = {row+1,col-1};
            temp[6] = {row+1,col};
            temp[7] = {row+1,col+1};
            for(int i = 0; i < sizeof(temp)/sizeof(*temp); i++)
	      bin[row][col].neighbor_bin_id.push_back(temp[i]);
    	  }
	}
      }
    }


    //
    //  simulate a number of time steps
    //
    double simulation_time = read_timer( );
	
    for( int step = 0; step < NSTEPS; step++ )
    {
	navg = 0;
        davg = 0.0;
	dmin = 1.0;

        // Reset particles in the bin to 0
        for(int i = 0; i < nbin_1d; i++) {
	  for(int j = 0; j < nbin_1d; j++) {
	    bin[i][j].particle_id.clear();
	  }
	}
 
        // Assign particles to bin and set acceleration to 0
        for(int i = 0; i < n; i++) {
	  int nx = floor(particles[i].x/bin_size);
	  int ny = floor(particles[i].y/bin_size);
	  if(nx == nbin_1d) nx--;
	  if(ny == nbin_1d) ny--;
	  bin[nx][ny].particle_id.push_back(i);
	  particles[i].ax = particles[i].ay = 0;
	}

        //
        //  iterate through bins to compute forces
        //
	for(int row = 0; row < nbin_1d; row++) {
	  for(int col = 0; col < nbin_1d; col++) {
	    bin_t temp_bin = bin[row][col];
	    // Iterate particles inside this bin to calculate force
	    for(int i = 0; i < temp_bin.particle_id.size(); i++) {
	      int id_this = temp_bin.particle_id.at(i);
	      // Interaction with particles inside this bin
	      for(int j = 0; j < temp_bin.particle_id.size(); j++) {
		if(j != i) {
		  int id_neip = temp_bin.particle_id.at(j);
		  apply_force(particles[id_this], particles[id_neip]);
		}
	      }
	      // Interaction with particles at neighboring bins
	      for(int nei = 0; nei < temp_bin.neighbor_bin_id.size(); nei++) {
		int nei_row = temp_bin.neighbor_bin_id.at(nei).at(0);
		int nei_col = temp_bin.neighbor_bin_id.at(nei).at(1);
		bin_t temp_nei = bin[nei_row][nei_col];
		for(int i = 0; i < temp_nei.particle_id.size(); i++) {
		  int id_neip = temp_nei.particle_id.at(i);
		  apply_force(particles[id_this], particles[id_neip]);
		}
	      }		
	    }
	  }
	}

        //
        //  move particles
        //
        for( int i = 0; i < n; i++ ) 
            move( particles[i] );		

        if( find_option( argc, argv, "-no" ) == -1 )
        {
          //
          // Computing statistical data
          //
          if (navg) {
            absavg +=  davg/navg;
            nabsavg++;
          }
          if (dmin < absmin) absmin = dmin;
		
          //
          //  save if necessary
          //
          if( fsave && (step%SAVEFREQ) == 0 )
              save( fsave, n, particles );
        }
    }
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "n = %d, simulation time = %g seconds", n, simulation_time);
    //
    // Printing summary data
    //
    if( fsum) 
        fprintf(fsum,"%d %g\n",n,simulation_time);
 
    //
    // Clearing space
    //
    if( fsum )
        fclose( fsum );    
    free( particles );
    if( fsave )
        fclose( fsave );
    
    return 0;
}
