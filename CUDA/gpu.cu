#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256

extern double size;
//
//  benchmarking program
//

__global__ void clear_bin_gpu(int nbin_1d, int* d_bin)
{
  size_t tidx = threadIdx.x + blockIdx.x*blockDim.x;
  size_t tidy = threadIdx.y + blockIdx.y*blockDim.y;

  if(tidx < nbin_1d && tidy < nbin_1d) {
    d_bin[tidx*nbin_1d + tidy] = -1;
  }
}

__global__ void assign_particle_gpu(int n, particle_t* d_particles, double bin_size, int nbin_1d, int* d_bin, int* d_particle_chain)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid < n) {
    particle_t* p_temp = &d_particles[tid];
    int nx = floor(p_temp->x/bin_size);
    int ny = floor(p_temp->y/bin_size);
    if(nx == nbin_1d) nx--;
    if(ny == nbin_1d) ny--;
    p_temp->ax = p_temp->ay = 0;
    d_particle_chain[tid] = atomicExch(&d_bin[nx*nbin_1d + ny], tid);
  }
}

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__global__ void compute_forces_gpu(int n, particle_t* d_particles, double bin_size, int nbin_1d, int* d_bin, int* d_particle_chain)
{
  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t* p_temp = &d_particles[tid];

  int nx = floor(p_temp->x/bin_size);
  int ny = floor(p_temp->y/bin_size);
  if(nx == nbin_1d) nx--;
  if(ny == nbin_1d) ny--;

  // Iterate through the particles in the same bin
  for(int i = d_bin[nx*nbin_1d + ny]; i != -1; i = d_particle_chain[i])
    if(i != tid) apply_force_gpu(*p_temp, d_particles[i]);
  // Iterate through the particles in neighboring bins
  if( nx == 0 && ny == 0 ) {
    for(int i = d_bin[(nx+1)*nbin_1d + ny]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[nx*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx+1)*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
  } else if( nx == 0 && ny == nbin_1d-1 ) {
    for(int i = d_bin[(nx+1)*nbin_1d + ny]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[nx*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx+1)*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
  } else if( nx == nbin_1d-1 && ny == 0 ) {
    for(int i = d_bin[(nx-1)*nbin_1d + ny]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[nx*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx-1)*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
  } else if(nx == nbin_1d-1 && ny == nbin_1d-1 ) {
    for(int i = d_bin[nx*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx-1)*nbin_1d + ny]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx-1)*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
  } else if(nx == 0 ) {
    for(int i = d_bin[nx*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[nx*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx+1)*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx+1)*nbin_1d + ny]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx+1)*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
  } else if(nx == nbin_1d-1 ) {
    for(int i = d_bin[nx*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[nx*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx-1)*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx-1)*nbin_1d + ny]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx-1)*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
  } else if(ny == 0 ) {
    for(int i = d_bin[(nx+1)*nbin_1d + ny]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx-1)*nbin_1d + ny]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx-1)*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[nx*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx+1)*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
  } else if(ny == nbin_1d-1 ) {
    for(int i = d_bin[(nx+1)*nbin_1d + ny]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx-1)*nbin_1d + ny]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx-1)*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[nx*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx+1)*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
   } else {
    for(int i = d_bin[nx*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[nx*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx-1)*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx-1)*nbin_1d + ny]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx-1)*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx+1)*nbin_1d + ny-1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx+1)*nbin_1d + ny]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
    for(int i = d_bin[(nx+1)*nbin_1d + ny+1]; i != -1; i = d_particle_chain[i]) apply_force_gpu(*p_temp, d_particles[i]);
  }

}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}


int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;

    // GPU bin structure
    int *d_particle_chain;
    int *d_bin;

    //int particle_blks = ceil(n/NUM_THREADS);
    int particle_blks = (n + NUM_THREADS - 1)/NUM_THREADS;

    double bin_size = 2*cutoff;
    double area_size = sqrt( density * n );
    int nbin_1d = ceil(1.0*area_size/bin_size);
    dim3 bin_threads(sqrt(NUM_THREADS), sqrt(NUM_THREADS));
    int bin_blk = ceil(1.0*nbin_1d/sqrt(NUM_THREADS));
    dim3 bin_blks(bin_blk, bin_blk); 

    hipMalloc((void **) &d_particle_chain, n * sizeof(int));
    hipMalloc((void **) &d_bin, nbin_1d*nbin_1d*sizeof(int)); 

    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        // Clear bins and assign particles to corresponding bin, reset acceleration
        clear_bin_gpu <<< bin_blks, bin_threads >>> (nbin_1d, d_bin);
	assign_particle_gpu <<< particle_blks, NUM_THREADS >>> (n, d_particles, bin_size, nbin_1d, d_bin, d_particle_chain);

        //
        //  compute forces
        //
	compute_forces_gpu <<< particle_blks, NUM_THREADS >>> (n, d_particles, bin_size, nbin_1d, d_bin, d_particle_chain);

        //
        //  move particles
        //
	move_gpu <<< particle_blks, NUM_THREADS >>> (d_particles, n, size);

        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
	    // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
	}
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
